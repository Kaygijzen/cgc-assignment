#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_GLOBAL __global__
#else
#define CUDA_GLOBAL
#endif

#include <hip/hip_runtime.h>
#include "module.h"
#include <stdio.h>
#include <math.h>

__device__ float calculate_distance(float avg, float item) {
	float diff = (avg - item);
	return diff * diff;
}

// THIS FUNCTION IS TENTATIVE AND NOT IMPLEMENTED
__global__ void cluster_id_kernel(
	int num_rows,
	int num_cols,
	const int* row_labels,
	const int* col_labels,
	int row_displacement,
	int* cluster_ids
) {
	int j = blockDim.x * blockIdx.x + threadIdx.x; 
	int tid = threadIdx.x;

	if (j < num_cols) {
		for (int i = row_displacement; i < num_rows + row_displacement; i++) {
			cluster_ids[i * num_cols + j] = row_labels[i] * col_labels[j];
		}
	}
}

// THIS FUNCTION IS TENTATIVE AND NOT IMPLEMENTED
void call_cluster_id_kernel(
	int num_rows,
	int num_cols,
	int num_row_labels,
	int num_col_labels,
	const float* matrix,
	const int* row_labels,
	const int* col_labels,
	int row_displacement,
	int* cluster_ids) {

	int N = num_cols;

	// Block size and number calculation
	int blockSize = 1024;
  int numBlocks = (N + blockSize - 1) / blockSize;

	// Allocate memory for data on device
	float *d_matrix;
	hipMalloc(&d_matrix, (num_cols*num_rows)*sizeof(float));
	int *d_cluster_ids;
	hipMalloc(&d_cluster_ids, (num_cols*num_rows)*sizeof(int));
	int *d_col_labels;
	hipMalloc(&d_col_labels, num_cols*sizeof(int));
	int *d_row_labels;
	hipMalloc(&d_row_labels, num_rows*sizeof(int));

	// Copy data to device
	hipMemcpy(d_row_labels, row_labels, (num_rows)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_col_labels, col_labels, (num_cols)*sizeof(int), hipMemcpyHostToDevice);

	cluster_id_kernel <<< numBlocks, blockSize >>>(
		num_rows,
		num_cols,
		d_row_labels,
		d_col_labels,
		row_displacement,
		d_cluster_ids);

	// Copy result from device to host
	hipMemcpy(cluster_ids, d_cluster_ids, (num_cols*num_rows)*sizeof(int), hipMemcpyDeviceToHost);
	
	// Free allocated memory
	hipFree(d_cluster_ids);
	hipFree(d_col_labels);
	hipFree(d_row_labels);
}

// THIS FUNCTION IS TENTATIVE AND NOT IMPLEMENTED
__global__ void cluster_sum_size_kernel(
	int num_rows,
	int num_cols,
	const float* matrix,
	const int* cluster_ids,
	double* cluster_sum,
	int* cluster_size,
	int row_displacement,
	int N
) {
	int id = blockDim.x * blockIdx.x + threadIdx.x; 
	int tid = threadIdx.x;

	if (id < (N)) {
		for (int i = row_displacement; i < num_rows + row_displacement; i++) {
			for (int j = 0; j < num_cols; j++) {
				int cluster_id = cluster_ids[i * num_cols + j];
				if (id == cluster_id) {
					cluster_sum[cluster_id] += matrix[cluster_ids[i * num_cols + j]];
					cluster_size[cluster_id] += 1;
				}
			}
		}
	}
};

// THIS FUNCTION IS TENTATIVE AND NOT IMPLEMENTED
void call_cluster_sum_size_kernel(
	int num_rows,
	int num_cols,
	int num_row_labels,
	int num_col_labels,
	const float* matrix,
	int* cluster_ids,
	int row_displacement,
	double* cluster_sum,
	int* cluster_size) {

	int N = num_col_labels * num_row_labels;

	// Block size and number calculation
	int blockSize = 1024;
  int numBlocks = (N + blockSize - 1) / blockSize;

	// Allocate memory for data on device
	float *d_matrix;
	hipMalloc(&d_matrix, (num_cols*num_rows)*sizeof(float));
	int *d_cluster_ids;
	hipMalloc(&d_cluster_ids, (num_cols*num_rows)*sizeof(int));
	double *d_cluster_sum;
	hipMalloc(&d_cluster_sum, (num_row_labels*num_col_labels)*sizeof(double));
	int *d_cluster_size;
	hipMalloc(&d_cluster_size, (num_row_labels*num_col_labels)*sizeof(int));

	// Copy data to device
	hipMemcpy(d_matrix, matrix, (num_cols*num_rows)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_ids, cluster_ids, (num_cols*num_rows)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_sum, cluster_sum, (num_col_labels*num_row_labels)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_size, cluster_size, (num_col_labels*num_row_labels)*sizeof(int), hipMemcpyHostToDevice);
	
	cluster_sum_size_kernel <<< numBlocks, blockSize >>>(
		num_rows,
		num_cols,
		d_matrix,
		d_cluster_ids,
		d_cluster_sum,
		d_cluster_size,
		row_displacement,
		(num_row_labels * num_col_labels));

	// Copy results from device to host
	hipMemcpy(cluster_sum, d_cluster_sum, (num_row_labels*num_col_labels)*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(cluster_size, d_cluster_size, (num_row_labels*num_col_labels)*sizeof(int), hipMemcpyDeviceToHost);

	// Free allocated memory
	hipFree(d_matrix);
	hipFree(d_cluster_ids);
	hipFree(d_cluster_sum);
	hipFree(d_cluster_size);
}

__global__ void calculate_block_distance_row(
	const float* matrix, 
	int i,
	int row_label,
	const int* col_labels,
	const float* cluster_avg,
	double* dist_per_block,
	int num_cols,
	int num_col_labels) 
{
	__shared__ double s_total_dist[1024];

	int j = blockDim.x * blockIdx.x + threadIdx.x; 
	int tid = threadIdx.x;

	if (j < num_cols) {
		float item = matrix[i * num_cols + j];

		int col_label = col_labels[j];

		float y = cluster_avg[row_label * num_col_labels + col_label];

		s_total_dist[tid] = calculate_distance(y, item);
	}

	__syncthreads();

	// do reduction in shared mem
	for (int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			s_total_dist[tid] += s_total_dist[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		dist_per_block[blockIdx.x] = s_total_dist[0];
	}
}

std::pair<int, double> call_update_row_labels_kernel(
    int num_rows,
    int num_cols,
    int num_row_labels,
    int num_col_labels,
    const float* matrix,
    int* row_labels,
    const int* col_labels,
    const float* cluster_avg,
    int displacement,
	int num_rows_recv) {
		
    int N = num_cols;

	// Block size and number calculation
	int blockSize = 1024;
  	int numBlocks = (N + blockSize - 1) / blockSize;
	
	// Number of bytes to allocate for numBlocks
	size_t bytes = numBlocks*sizeof(double);

	// Allocate memory on host
	double *dist_blocks = (double*)malloc(bytes);

	// Allocate memory on device
	double *d_dist_blocks;
	hipMalloc(&d_dist_blocks, bytes);
	float *d_matrix;
	hipMalloc(&d_matrix, (num_rows*num_cols)*sizeof(float));
	float *d_cluster_avg;
	hipMalloc(&d_cluster_avg, (num_row_labels*num_col_labels)*sizeof(float));
	int *d_col_labels;
	hipMalloc(&d_col_labels, num_cols*sizeof(int));

	// Copy data to device
	hipMemcpy(d_matrix, matrix, (num_rows*num_cols)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_avg, cluster_avg, (num_row_labels*num_col_labels)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_col_labels, col_labels, num_cols*sizeof(int), hipMemcpyHostToDevice);

    int num_updated = 0;
    double total_dist = 0;

    for (int i = 0; i < num_rows_recv; i++) {
        int best_label = -1;
        double best_dist = INFINITY;
        int displaced_i = i + displacement;

        for (int k = 0; k < num_row_labels; k++) {
            double dist = 0;

			calculate_block_distance_row <<< numBlocks, blockSize >>>(
				d_matrix,
				displaced_i,
				k,
				d_col_labels,
				d_cluster_avg,
				d_dist_blocks, 
				num_cols,
				num_col_labels);

			// Copy result from device to host
			hipMemcpy(dist_blocks, d_dist_blocks, bytes, hipMemcpyDeviceToHost);

			// Reduce result by summing all block results
			for (int x = 0; x < numBlocks; x++) {
				dist += dist_blocks[x];
			}

			if (dist < best_dist) {
				best_dist = dist;
				best_label = k;
			}
        }

        if (row_labels[i] != best_label) {
            row_labels[i] = best_label;
            num_updated++;
        }

        total_dist += best_dist;
    }

	// Free allocated memory
	hipFree(d_dist_blocks);
	hipFree(d_matrix);
	hipFree(d_col_labels);
	free(dist_blocks);

	return {num_updated, total_dist};
}


__global__ void col_labels_iteration(
	const float* matrix, 
	int* col_labels,
	const int* row_labels,
	const float* cluster_avg,
	int* num_updated_per_block,
	double* total_dist_per_block,
	int num_cols,
	int num_rows,
	int num_col_labels,
	int num_cols_recv,
	int displacement) {

	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;

	__shared__ int s_num_updated[1024];
	__shared__ double s_total_dist[1024];
	s_num_updated[tid] = 0;
	s_total_dist[tid] = 0.0;

	__syncthreads();

    if (j < num_cols_recv) {
        int best_label = -1;
        double best_dist = INFINITY;

        for (int k = 0; k < num_col_labels; k++) {
            double dist = 0;

            for (int i = 0; i < num_rows; i++) {
                auto item = matrix[i * num_cols + j + displacement];

                auto row_label = row_labels[i];
                auto col_label = k;
                auto y = cluster_avg[row_label * num_col_labels + col_label];

                dist += calculate_distance(y, item);
            }

            if (dist < best_dist) {
                best_dist = dist;
                best_label = k;
            }
        }

        if (col_labels[j] != best_label) {
            col_labels[j] = best_label;
            s_num_updated[tid]++;
        }

        s_total_dist[tid] += best_dist;
    }

	__syncthreads();

	// do reduction in shared mem
	for (int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			s_num_updated[tid] += s_num_updated[tid + s];
			s_total_dist[tid] += s_total_dist[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		num_updated_per_block[blockIdx.x] = s_num_updated[0];
		total_dist_per_block[blockIdx.x] = s_total_dist[0];
	}
}

std::pair<int, double> call_update_col_labels_kernel(
	int num_rows,
	int num_cols,
	int num_row_labels,
	int num_col_labels,
	const float* matrix,
	const int* row_labels,
	int* col_labels,
	const float* cluster_avg,
	int displacement,
	int num_cols_recv) {

	int N = num_cols_recv;

	// Block size and number calculation
	int blockSize = 1024;
    int numBlocks = (N + blockSize - 1) / blockSize;

	// Allocate memory on host
	int *num_updated_per_block = (int*)malloc(numBlocks*sizeof(int));
	double *total_dist_per_block = (double*)malloc(numBlocks*sizeof(double));

	// Allocate memory for data on device
	float *d_matrix;
	hipMalloc(&d_matrix, (num_cols*num_rows)*sizeof(float));
	float *d_cluster_avg;
	hipMalloc(&d_cluster_avg, (num_row_labels*num_col_labels)*sizeof(float));
	int *d_col_labels;
	hipMalloc(&d_col_labels, num_cols_recv*sizeof(int));
	int *d_row_labels;
	hipMalloc(&d_row_labels, num_rows*sizeof(int));
	int *d_num_updated_per_block;
	hipMalloc(&d_num_updated_per_block, numBlocks*sizeof(int));
	double *d_total_dist_per_block;
	hipMalloc(&d_total_dist_per_block, numBlocks*sizeof(double));

	// Copy data to device
	hipMemcpy(d_matrix, matrix, (num_cols*num_rows)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_avg, cluster_avg, (num_row_labels*num_col_labels)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_col_labels, col_labels, num_cols_recv*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_labels, row_labels, num_rows*sizeof(int), hipMemcpyHostToDevice);

	// Call kernel
	col_labels_iteration <<< numBlocks, blockSize >>>(
		d_matrix, 
		d_col_labels,
		d_row_labels,
		d_cluster_avg,
		d_num_updated_per_block,
		d_total_dist_per_block,
		num_cols,
		num_rows,
		num_col_labels,
		num_cols_recv,
		displacement);

	// Copy results from device to host
	hipMemcpy(col_labels, d_col_labels, num_cols_recv*sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(num_updated_per_block, d_num_updated_per_block, numBlocks*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(total_dist_per_block, d_total_dist_per_block, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

	int num_updated = 0;
	double total_dist = 0;

	// Reduce result by summing all block results
	for (int i = 0; i < numBlocks; i++) {
		num_updated += num_updated_per_block[i];
		total_dist += total_dist_per_block[i];
	}

	// Free allocated memory
	hipFree(d_matrix);
	hipFree(d_cluster_avg);
	hipFree(d_col_labels);
	hipFree(d_row_labels);

	return {num_updated, total_dist};
}
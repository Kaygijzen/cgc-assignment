#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_GLOBAL __global__
#else
#define CUDA_GLOBAL
#endif

#include <hip/hip_runtime.h>
#include "module.h"
#include <stdio.h>
#include <math.h>

__global__ void block_dist_row_labels(
	const float* matrix, 
	int i,
	int k,
	const int* col_labels,
	const float* cluster_avg,
	double* dist_array,
	int num_cols,
	int num_col_labels) 
{
	__shared__ double sdata[1024];

	int j = blockDim.x * blockIdx.x + threadIdx.x; 
	int tid = threadIdx.x;

	if (j < num_cols) {
		float item = matrix[i * num_cols + j];

		int row_label = k;
		int col_label = col_labels[j];

		float y = cluster_avg[row_label * num_col_labels + col_label];

		sdata[tid] = (y - item) * (y - item);
	}

	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		dist_array[blockIdx.x] = sdata[0];
	}
}

std::pair<int, double> best_label_row(
	int num_row_labels,
	int num_col_labels,
	int num_rows,
	int num_cols,
	const float* matrix,
	const float* cluster_avg,
	int i,
	const int* col_labels) {
	int N = num_cols;

	// Block size and number calculation
	int blockSize = 1024;
  int numBlocks = (N + blockSize - 1) / blockSize;
	
	// Number of bytes to allocate for numBlocks
	size_t bytes = numBlocks*sizeof(double);

	// Allocate memory on host
	double *dist_blocks = (double*)malloc(bytes);

	// Allocate memory on device
	double *d_dist_blocks;
	hipMalloc(&d_dist_blocks, bytes);
	float *d_matrix;
	hipMalloc(&d_matrix, (num_cols*num_rows)*sizeof(float));
	float *d_cluster_avg;
	hipMalloc(&d_cluster_avg, (num_row_labels*num_col_labels)*sizeof(float));
	int *d_col_labels;
	hipMalloc(&d_col_labels, num_cols*sizeof(int));

	// Copy data to device
	hipMemcpy(d_matrix, matrix, (num_cols*num_rows)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_avg, cluster_avg, (num_row_labels*num_col_labels)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_col_labels, col_labels, num_cols*sizeof(int), hipMemcpyHostToDevice);

	int best_label = -1;
	double best_dist = INFINITY;

	for (int k = 0; k < num_row_labels; k++) {
		double dist = 0;

		block_dist_row_labels<<< numBlocks, blockSize >>>(
			d_matrix,
			i,
			k,
			d_col_labels,
			d_cluster_avg,
			d_dist_blocks, 
			num_cols,
			num_col_labels);

		hipDeviceSynchronize();

		// Copy result from device to host
		hipMemcpy(dist_blocks, d_dist_blocks, bytes, hipMemcpyDeviceToHost);

		// Reduce result by summing all block results
		double sum = 0;
		for (int x = 0; x < numBlocks; x++) {
			sum += dist_blocks[x];
		}

		dist = sum;

		if (dist < best_dist) {
			best_dist = dist;
			best_label = k;
		}
	}

	// Free allocated memory
	hipFree(d_dist_blocks);
	hipFree(d_matrix);
	hipFree(d_col_labels);
	free(dist_blocks);

	return {best_label, best_dist};
}
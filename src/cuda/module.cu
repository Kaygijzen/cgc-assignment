#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_GLOBAL __global__
#else
#define CUDA_GLOBAL
#endif

#include <hip/hip_runtime.h>
#include "module.h"
#include <stdio.h>
#include <math.h>

__device__ float calculate_distance(float avg, float item) {
	float diff = (avg - item);
	return diff * diff;
}

__global__ void cluster_id_kernel(
	int num_rows,
	int num_cols,
	const int* row_labels,
	const int* col_labels,
	int* cluster_ids,
	int num_col_labels,
	int row_displacement
) {
	int j = blockDim.x * blockIdx.x + threadIdx.x; 

	if (j < num_cols) {
		for (int i = 0; i < num_rows; i++) {
			int row_label = row_labels[i + row_displacement];
			int col_label = col_labels[j];
			cluster_ids[i * num_cols + j] = row_label * num_col_labels + col_label;
		}
	}
}

void call_cluster_id_kernel(
	int num_rows,
	int num_cols,
	int num_col_labels,
	const int* row_labels,
	const int* col_labels,
	int* cluster_ids,
	int row_displacement,
	int num_rows_recv) {

	// Block size and number calculation
	int blockSize = 1024;
  	int numBlocks = (num_cols + blockSize - 1) / blockSize;

	// Allocate memory for data on device
	int *d_row_labels;
	hipMalloc(&d_row_labels, num_rows*sizeof(int));
	int *d_col_labels;
	hipMalloc(&d_col_labels, num_cols*sizeof(int));
	int *d_cluster_ids;
	hipMalloc(&d_cluster_ids, (num_rows_recv*num_cols)*sizeof(int));

	// Copy data to device
	hipMemcpy(d_row_labels, row_labels, num_rows*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_col_labels, col_labels, num_cols*sizeof(int), hipMemcpyHostToDevice);
	
	cluster_id_kernel <<< numBlocks, blockSize >>>(
		num_rows_recv,
		num_cols,
		d_row_labels,
		d_col_labels,
		d_cluster_ids,
		num_col_labels,
		row_displacement);

	// Copy results from device to host
	hipMemcpy(cluster_ids, d_cluster_ids, num_rows_recv*num_cols*sizeof(int), hipMemcpyDeviceToHost);

	// Free allocated memory
	hipFree(d_row_labels);
	hipFree(d_col_labels);
	hipFree(d_cluster_ids);
}

__global__ void calculate_cluster_avg(
	int num_row_labels,
	int num_col_labels,
	double* cluster_sum,
	int* cluster_size,
	float* cluster_avg) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < num_row_labels && j < num_col_labels) {
		cluster_avg[i * num_col_labels + j] = float(cluster_sum[i * num_col_labels + j]) / float(cluster_size[i * num_col_labels + j]);   
    }
}

void call_cluster_average_kernel(
	int num_row_labels,
	int num_col_labels,
	double* cluster_sum,
	int* cluster_size,
	float* cluster_avg) {

	int N = num_col_labels;
	int num_clusters = num_row_labels * num_col_labels;

	// Block size and number calculation
	dim3 blockSize(32, 32);
    dim3 numBlocks((N + blockSize.x - 1) / blockSize.x, (N+blockSize.y -1) / blockSize.y);

	// Allocate memory on device
	double *d_cluster_sum;
	hipMalloc(&d_cluster_sum, num_clusters*sizeof(double));
	int *d_cluster_size;
	hipMalloc(&d_cluster_size, num_clusters*sizeof(int));
	float *d_cluster_avg;
	hipMalloc(&d_cluster_avg, num_clusters*sizeof(float));

	// Copy data to device
	hipMemcpy(d_cluster_sum, cluster_sum, num_clusters*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_size, cluster_size, num_clusters*sizeof(int), hipMemcpyHostToDevice);

	calculate_cluster_avg <<< numBlocks, blockSize >>>(
		num_row_labels,
		num_col_labels,
		d_cluster_sum,
		d_cluster_size,
		d_cluster_avg); 

	// Copy result from device to host
	hipMemcpy(cluster_avg, d_cluster_avg, num_clusters*sizeof(float), hipMemcpyDeviceToHost);

	// Free allocated memory
	hipFree(d_cluster_sum);
	hipFree(d_cluster_size);
	hipFree(d_cluster_avg);
}

__global__ void calculate_block_distance_row(
	const float* matrix, 
	int i,
	int row_label,
	const int* col_labels,
	const float* cluster_avg,
	double* dist_per_block,
	int num_cols,
	int num_col_labels) 
{
	__shared__ double s_total_dist[1024];

	int j = blockDim.x * blockIdx.x + threadIdx.x; 
	int tid = threadIdx.x;

	if (j < num_cols) {
		float item = matrix[i * num_cols + j];

		int col_label = col_labels[j];

		float y = cluster_avg[row_label * num_col_labels + col_label];

		s_total_dist[tid] = calculate_distance(y, item);
	}

	__syncthreads();

	// do reduction in shared mem
	for (int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			s_total_dist[tid] += s_total_dist[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		dist_per_block[blockIdx.x] = s_total_dist[0];
	}
}

std::pair<int, double> call_update_row_labels_kernel(
    int num_rows,
    int num_cols,
    int num_row_labels,
    int num_col_labels,
    const float* matrix,
    int* row_labels,
    const int* col_labels,
    const float* cluster_avg,
    int displacement,
	int num_rows_recv) {
		
    int N = num_cols;

	// Block size and number calculation
	int blockSize = 1024;
  	int numBlocks = (N + blockSize - 1) / blockSize;
	
	// Number of bytes to allocate for numBlocks
	size_t bytes = numBlocks*sizeof(double);

	// Allocate memory on host
	double *dist_blocks = (double*)malloc(bytes);

	// Allocate memory on device
	double *d_dist_blocks;
	hipMalloc(&d_dist_blocks, bytes);
	float *d_matrix;
	hipMalloc(&d_matrix, (num_rows*num_cols)*sizeof(float));
	float *d_cluster_avg;
	hipMalloc(&d_cluster_avg, (num_row_labels*num_col_labels)*sizeof(float));
	int *d_col_labels;
	hipMalloc(&d_col_labels, num_cols*sizeof(int));

	// Copy data to device
	hipMemcpy(d_matrix, matrix, (num_rows*num_cols)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_avg, cluster_avg, (num_row_labels*num_col_labels)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_col_labels, col_labels, num_cols*sizeof(int), hipMemcpyHostToDevice);

    int num_updated = 0;
    double total_dist = 0;

    for (int i = 0; i < num_rows_recv; i++) {
        int best_label = -1;
        double best_dist = INFINITY;
        int displaced_i = i + displacement;

        for (int k = 0; k < num_row_labels; k++) {
            double dist = 0;

			calculate_block_distance_row <<< numBlocks, blockSize >>>(
				d_matrix,
				displaced_i,
				k,
				d_col_labels,
				d_cluster_avg,
				d_dist_blocks, 
				num_cols,
				num_col_labels);

			// Copy result from device to host
			hipMemcpy(dist_blocks, d_dist_blocks, bytes, hipMemcpyDeviceToHost);

			// Reduce result by summing all block results
			for (int x = 0; x < numBlocks; x++) {
				dist += dist_blocks[x];
			}

			if (dist < best_dist) {
				best_dist = dist;
				best_label = k;
			}
        }

        if (row_labels[i] != best_label) {
            row_labels[i] = best_label;
            num_updated++;
        }

        total_dist += best_dist;
    }

	// Free allocated memory
	hipFree(d_dist_blocks);
	hipFree(d_matrix);
	hipFree(d_col_labels);
	free(dist_blocks);

	return {num_updated, total_dist};
}


__global__ void col_labels_iteration(
	const float* matrix, 
	int* col_labels,
	const int* row_labels,
	const float* cluster_avg,
	int* num_updated_per_block,
	double* total_dist_per_block,
	int num_cols,
	int num_rows,
	int num_col_labels,
	int num_cols_recv,
	int displacement) {

	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;

	__shared__ int s_num_updated[1024];
	__shared__ double s_total_dist[1024];
	s_num_updated[tid] = 0;
	s_total_dist[tid] = 0.0;

	__syncthreads();

    if (j < num_cols_recv) {
        int best_label = -1;
        double best_dist = INFINITY;

        for (int k = 0; k < num_col_labels; k++) {
            double dist = 0;

            for (int i = 0; i < num_rows; i++) {
                auto item = matrix[i * num_cols + j + displacement];

                auto row_label = row_labels[i];
                auto col_label = k;
                auto y = cluster_avg[row_label * num_col_labels + col_label];

                dist += calculate_distance(y, item);
            }

            if (dist < best_dist) {
                best_dist = dist;
                best_label = k;
            }
        }

        if (col_labels[j] != best_label) {
            col_labels[j] = best_label;
            s_num_updated[tid]++;
        }

        s_total_dist[tid] += best_dist;
    }

	__syncthreads();

	// do reduction in shared mem
	for (int s=1; s < blockDim.x; s *= 2) {
		if (tid % (2*s) == 0) {
			s_num_updated[tid] += s_num_updated[tid + s];
			s_total_dist[tid] += s_total_dist[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		num_updated_per_block[blockIdx.x] = s_num_updated[0];
		total_dist_per_block[blockIdx.x] = s_total_dist[0];
	}
}

std::pair<int, double> call_update_col_labels_kernel(
	int num_rows,
	int num_cols,
	int num_row_labels,
	int num_col_labels,
	const float* matrix,
	const int* row_labels,
	int* col_labels,
	const float* cluster_avg,
	int displacement,
	int num_cols_recv) {

	int N = num_cols_recv;

	// Block size and number calculation
	int blockSize = 1024;
    int numBlocks = (N + blockSize - 1) / blockSize;

	// Allocate memory on host
	int *num_updated_per_block = (int*)malloc(numBlocks*sizeof(int));
	double *total_dist_per_block = (double*)malloc(numBlocks*sizeof(double));

	// Allocate memory for data on device
	float *d_matrix;
	hipMalloc(&d_matrix, (num_cols*num_rows)*sizeof(float));
	float *d_cluster_avg;
	hipMalloc(&d_cluster_avg, (num_row_labels*num_col_labels)*sizeof(float));
	int *d_col_labels;
	hipMalloc(&d_col_labels, num_cols_recv*sizeof(int));
	int *d_row_labels;
	hipMalloc(&d_row_labels, num_rows*sizeof(int));
	int *d_num_updated_per_block;
	hipMalloc(&d_num_updated_per_block, numBlocks*sizeof(int));
	double *d_total_dist_per_block;
	hipMalloc(&d_total_dist_per_block, numBlocks*sizeof(double));

	// Copy data to device
	hipMemcpy(d_matrix, matrix, (num_cols*num_rows)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster_avg, cluster_avg, (num_row_labels*num_col_labels)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_col_labels, col_labels, num_cols_recv*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_labels, row_labels, num_rows*sizeof(int), hipMemcpyHostToDevice);

	// Call kernel
	col_labels_iteration <<< numBlocks, blockSize >>>(
		d_matrix, 
		d_col_labels,
		d_row_labels,
		d_cluster_avg,
		d_num_updated_per_block,
		d_total_dist_per_block,
		num_cols,
		num_rows,
		num_col_labels,
		num_cols_recv,
		displacement);

	// Copy results from device to host
	hipMemcpy(col_labels, d_col_labels, num_cols_recv*sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(num_updated_per_block, d_num_updated_per_block, numBlocks*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(total_dist_per_block, d_total_dist_per_block, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

	int num_updated = 0;
	double total_dist = 0;

	// Reduce result by summing all block results
	for (int i = 0; i < numBlocks; i++) {
		num_updated += num_updated_per_block[i];
		total_dist += total_dist_per_block[i];
	}

	// Free allocated memory
	hipFree(d_matrix);
	hipFree(d_cluster_avg);
	hipFree(d_col_labels);
	hipFree(d_row_labels);

	return {num_updated, total_dist};
}